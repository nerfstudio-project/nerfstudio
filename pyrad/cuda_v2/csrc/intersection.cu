#include "hip/hip_runtime.h"
#include "include/helpers.h"


template <typename scalar_t>
inline __host__ __device__ void _ray_aabb_intersect(
    const scalar_t* rays_o,
    const scalar_t* rays_d,
    const scalar_t* aabb,
    scalar_t* near,
    scalar_t* far
) {
    // aabb is [xmin, ymin, zmin, xmax, ymax, zmax]
    scalar_t tmin = (aabb[0] - rays_o[0]) / rays_d[0];
    scalar_t tmax = (aabb[3] - rays_o[0]) / rays_d[0];
    if (tmin > tmax) __swap(tmin, tmax);

    scalar_t tymin = (aabb[1] - rays_o[1]) / rays_d[1];
    scalar_t tymax = (aabb[4] - rays_o[1]) / rays_d[1];
    if (tymin > tymax) __swap(tymin, tymax);

    if (tmin > tymax || tymin > tmax){
        *near = std::numeric_limits<scalar_t>::max();
        *far = std::numeric_limits<scalar_t>::max();
        return;
    }

    if (tymin > tmin) tmin = tymin;
    if (tymax < tmax) tmax = tymax;

    scalar_t tzmin = (aabb[2] - rays_o[2]) / rays_d[2];
    scalar_t tzmax = (aabb[5] - rays_o[2]) / rays_d[2];
    if (tzmin > tzmax) __swap(tzmin, tzmax);

    if (tmin > tzmax || tzmin > tmax){
        *near = std::numeric_limits<scalar_t>::max();
        *far = std::numeric_limits<scalar_t>::max();
        return;
    }

    if (tzmin > tmin) tmin = tzmin;
    if (tzmax < tmax) tmax = tzmax;

    *near = tmin;
    *far = tmax;
    return;
}


template <typename scalar_t>
__global__ void kernel_ray_aabb_intersect(
    const int N,
    const scalar_t* rays_o,
    const scalar_t* rays_d,
    const scalar_t* aabb,
    scalar_t* t_min,
    scalar_t* t_max
){
    // aabb is [xmin, ymin, zmin, xmax, ymax, zmax]
    CUDA_GET_THREAD_ID(thread_id, N);

    // locate
    rays_o += thread_id * 3;
    rays_d += thread_id * 3;
    t_min += thread_id;
    t_max += thread_id;
    
    _ray_aabb_intersect<scalar_t>(rays_o, rays_d, aabb, t_min, t_max);
    
    // The near distance prevents learning of camera-specific fudge 
    // right in front of the camera.
    scalar_t zero = static_cast<scalar_t>(0.f);
    *t_min = *t_min > zero ? *t_min : zero;

    return;
}

/**
 * @brief Ray AABB Test
 * 
 * @param rays_o Ray origins. Tensor with shape [N, 3]
 * @param rays_d Ray directions. Tensor with shape [N, 3]
 * @param aabb_min AABB min. Tensor with shape [3]
 * @param aabb_max AABB max. Tensor with shape [3]
 * @return std::vector<torch::Tensor> t_min and t_max with shape [N]
 */
std::vector<torch::Tensor> ray_aabb_intersect(
    const torch::Tensor rays_o,
    const torch::Tensor rays_d,
    const torch::Tensor aabb
) {
    DEVICE_GUARD(rays_o);
    CHECK_INPUT(rays_o);
    CHECK_INPUT(rays_d);
    CHECK_INPUT(aabb);
    const int N = rays_o.size(0);

    const int threads = 256;
    const int blocks = CUDA_N_BLOCKS_NEEDED(N, threads);

    torch::Tensor t_min = torch::empty({N}, rays_o.options());
    torch::Tensor t_max = torch::empty({N}, rays_o.options());

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
        rays_o.scalar_type(), "ray_aabb_intersect", 
        ([&] {
            kernel_ray_aabb_intersect<scalar_t><<<blocks, threads>>>(
                N,
                rays_o.data_ptr<scalar_t>(),
                rays_d.data_ptr<scalar_t>(),
                aabb.data_ptr<scalar_t>(),
                t_min.data_ptr<scalar_t>(),
                t_max.data_ptr<scalar_t>()
            );
        })
    );

    return {t_min, t_max};
}