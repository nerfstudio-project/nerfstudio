#include "hip/hip_runtime.h"
#include "utils.h"


template <typename scalar_t>
__global__ void composite_train_fw_kernel(
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> sigmas,
    const torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> rgbs,
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> deltas,
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> ts,
    const torch::PackedTensorAccessor32<int, 2, torch::RestrictPtrTraits> rays_a,
    const scalar_t T_threshold,
    torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> opacity,
    torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> depth,
    torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> rgb
){
    const int n = blockIdx.x * blockDim.x + threadIdx.x;
    if (n >= opacity.size(0)) return;

    const int ray_idx = rays_a[n][0], start_idx = rays_a[n][1], N_samples = rays_a[n][2];
    if (N_samples==0 || start_idx+N_samples>=sigmas.size(0)) return; // no hit

    // front to back compositing
    int samples = 0; scalar_t T = 1.0f;

    while (samples < N_samples) {
        const int s = start_idx + samples;
        const scalar_t a = 1.0f - __expf(-sigmas[s]*deltas[s]);
        const scalar_t w = a * T;

        rgb[ray_idx][0] += w*rgbs[s][0];
        rgb[ray_idx][1] += w*rgbs[s][1];
        rgb[ray_idx][2] += w*rgbs[s][2];
        depth[ray_idx] += w*ts[s];
        opacity[ray_idx] += w;
        T *= 1.0f-a;

        if (T <= T_threshold) break; // ray has enough opacity
        samples++;
    }
}


std::vector<torch::Tensor> composite_train_fw_cu(
    const torch::Tensor sigmas,
    const torch::Tensor rgbs,
    const torch::Tensor deltas,
    const torch::Tensor ts,
    const torch::Tensor rays_a,
    const float T_threshold
){
    const int N_rays = rays_a.size(0);

    auto opacity = torch::zeros({N_rays}, sigmas.options());
    auto depth = torch::zeros({N_rays}, sigmas.options());
    auto rgb = torch::zeros({N_rays, 3}, sigmas.options());

    const int threads = 256, blocks = (N_rays+threads-1)/threads;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(sigmas.type(), "composite_train_fw_cu", 
    ([&] {
        composite_train_fw_kernel<scalar_t><<<blocks, threads>>>(
            sigmas.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            rgbs.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>(),
            deltas.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            ts.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            rays_a.packed_accessor32<int, 2, torch::RestrictPtrTraits>(),
            T_threshold,
            opacity.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            depth.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            rgb.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>()
        );
    }));

    return {opacity, depth, rgb};
}


template <typename scalar_t>
__global__ void composite_train_bw_kernel(
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> dL_dopacity,
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> dL_ddepth,
    const torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> dL_drgb,
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> sigmas,
    const torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> rgbs,
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> deltas,
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> ts,
    const torch::PackedTensorAccessor32<int, 2, torch::RestrictPtrTraits> rays_a,
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> opacity,
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> depth,
    const torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> rgb,
    const scalar_t T_threshold,
    torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> dL_dsigmas,
    torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> dL_drgbs
){
    const int n = blockIdx.x * blockDim.x + threadIdx.x;
    if (n >= opacity.size(0)) return;

    const int ray_idx = rays_a[n][0], start_idx = rays_a[n][1], N_samples = rays_a[n][2];
    if (N_samples==0 || start_idx+N_samples>=sigmas.size(0)) return; // no hit
        
    // front to back compositing
    int samples = 0;
    scalar_t R = rgb[ray_idx][0], G = rgb[ray_idx][1], B = rgb[ray_idx][2];
    scalar_t O = opacity[ray_idx], D = depth[ray_idx];
    scalar_t T = 1.0f, r = 0.0f, g = 0.0f, b = 0.0f, op = 0.0f, t = 0.0f, d = 0.0f;

    while (samples < N_samples) {
        const int s = start_idx + samples;
        const scalar_t a = 1.0f - __expf(-sigmas[s]*deltas[s]);
        const scalar_t w = a * T;

        r += w*rgbs[s][0]; g += w*rgbs[s][1]; b += w*rgbs[s][2];
        d += w*ts[s];
        op += w;
        T *= 1.0f-a;

        // compute gradients by math...
        dL_drgbs[s][0] = dL_drgb[ray_idx][0]*w;
        dL_drgbs[s][1] = dL_drgb[ray_idx][1]*w;
        dL_drgbs[s][2] = dL_drgb[ray_idx][2]*w;

        dL_dsigmas[s] = deltas[s] * (
            dL_drgb[ray_idx][0]*(rgbs[s][0]*T-(R-r)) + 
            dL_drgb[ray_idx][1]*(rgbs[s][1]*T-(G-g)) + 
            dL_drgb[ray_idx][2]*(rgbs[s][2]*T-(B-b)) + 
            dL_dopacity[ray_idx]*(1-O) + 
            dL_ddepth[ray_idx]*(t*T-(D-d))
        );

        if (T <= T_threshold) break; // ray has enough opacity
        samples++;
    }
}


std::vector<torch::Tensor> composite_train_bw_cu(
    const torch::Tensor dL_dopacity,
    const torch::Tensor dL_ddepth,
    const torch::Tensor dL_drgb,
    const torch::Tensor sigmas,
    const torch::Tensor rgbs,
    const torch::Tensor deltas,
    const torch::Tensor ts,
    const torch::Tensor rays_a,
    const torch::Tensor opacity,
    const torch::Tensor depth,
    const torch::Tensor rgb,
    // const torch::Tensor rgb_bg,
    const float T_threshold
){
    const int N = sigmas.size(0), N_rays = rays_a.size(0);

    auto dL_dsigmas = torch::zeros({N}, sigmas.options());
    auto dL_drgbs = torch::zeros({N, 3}, sigmas.options());

    const int threads = 256, blocks = (N_rays+threads-1)/threads;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(sigmas.type(), "composite_train_bw_cu", 
    ([&] {
        composite_train_bw_kernel<scalar_t><<<blocks, threads>>>(
            dL_dopacity.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            dL_ddepth.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            dL_drgb.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>(),
            sigmas.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            rgbs.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>(),
            deltas.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            ts.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            rays_a.packed_accessor32<int, 2, torch::RestrictPtrTraits>(),
            opacity.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            depth.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            rgb.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>(),
            T_threshold,
            dL_dsigmas.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            dL_drgbs.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>()
        );
    }));

    return {dL_dsigmas, dL_drgbs};
}


template <typename scalar_t>
__global__ void composite_test_fw_kernel(
    const torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> sigmas,
    const torch::PackedTensorAccessor<scalar_t, 3, torch::RestrictPtrTraits, size_t> rgbs,
    const torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> deltas,
    const torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> ts,
    const torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> hits_t,
    torch::PackedTensorAccessor64<long, 1, torch::RestrictPtrTraits> alive_indices,
    const scalar_t T_threshold,
    const torch::PackedTensorAccessor32<int, 1, torch::RestrictPtrTraits> N_eff_samples,
    torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> opacity,
    torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> depth,
    torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> rgb
){
    const int n = blockIdx.x * blockDim.x + threadIdx.x;
    if (n >= alive_indices.size(0)) return;

    if (N_eff_samples[n]==0){ // no hit
        alive_indices[n] = -1;
        return;
    }

    const size_t r = alive_indices[n]; // ray index

    // front to back compositing
    int s = 0; scalar_t T = 1-opacity[r];

    while (s < N_eff_samples[n]) {
        const scalar_t a = 1.0f - __expf(-sigmas[n][s]*deltas[n][s]);
        const scalar_t w = a * T;

        rgb[r][0] += w*rgbs[n][s][0];
        rgb[r][1] += w*rgbs[n][s][1];
        rgb[r][2] += w*rgbs[n][s][2];
        depth[r] += w*ts[n][s];
        opacity[r] += w;
        T *= 1.0f-a;

        if (T <= T_threshold){ // ray has enough opacity
            alive_indices[n] = -1;
            break;
        }
        s++;
    }
}


void composite_test_fw_cu(
    const torch::Tensor sigmas,
    const torch::Tensor rgbs,
    const torch::Tensor deltas,
    const torch::Tensor ts,
    const torch::Tensor hits_t,
    torch::Tensor alive_indices,
    const float T_threshold,
    const torch::Tensor N_eff_samples,
    torch::Tensor opacity,
    torch::Tensor depth,
    torch::Tensor rgb
){
    const int N_rays = alive_indices.size(0);

    const int threads = 256, blocks = (N_rays+threads-1)/threads;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(sigmas.type(), "composite_test_fw_cu", 
    ([&] {
        composite_test_fw_kernel<scalar_t><<<blocks, threads>>>(
            sigmas.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>(),
            rgbs.packed_accessor<scalar_t, 3, torch::RestrictPtrTraits, size_t>(),
            deltas.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>(),
            ts.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>(),
            hits_t.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>(),
            alive_indices.packed_accessor64<long, 1, torch::RestrictPtrTraits>(),
            T_threshold,
            N_eff_samples.packed_accessor32<int, 1, torch::RestrictPtrTraits>(),
            opacity.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            depth.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            rgb.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>()
        );
    }));
}